#include "hip/hip_runtime.h"
#include "LBM_2D_NS_cuKernel.h"

__constant__ unsigned int mirrorNS[13] = {0,1,4,3,2,8,7,6,5,9,12,11,10};
__constant__ unsigned int mirrorEW[13] = {0,3,2,1,4,6,5,8,7,11,10,9,12};

__constant__ int xx[4] = {-1,1,0,0};
__constant__ int yy[4] = {0,0,-1,1};
__constant__ unsigned	int ii[4][4] = {{1,5,8,9},{3,6,7,11},{2,5,6,10},{4,7,8,12}};
__constant__ unsigned	int inv[4][4] = {{3,6,7,11},{1,5,8,9},{4,7,8,12},{2,5,6,10}};

__device__ int errEq = 0;

////////////////////////////////////////////////////////////////////////////////
// INLINE
////////////////////////////////////////////////////////////////////////////////

// equilibrium functions for f

__device__ inline double f0(double rho, double u, double v, double T)
{
	//rest particle equilibrium function
	return (rho/4.0)*(4 + 10*pow(T,2) + pow(u,4) - 5*pow(v,2) + pow(v,4) + 10*T*(-1 + pow(u,2) + pow(v,2)) + pow(u,2)*(-5 + 4*pow(v,2)));
}

__device__ inline double f1(double rho, double u, double v, double T)
{
	return (rho/6.0)*(-6*pow(T,2) - u*(1 + u)*(-4 + pow(u,2) + 3*pow(v,2)) - T*(-4 + 6*u + 9*pow(u,2) + 3*pow(v,2)));
}

__device__ inline double f5(double rho, double u, double v, double T)
{
	return (rho/4.0)*((T + u + pow(u,2))*(T + v + pow(v,2)));
}

__device__ inline double f9(double rho, double u, double v, double T)
{
	return (rho/24.0)*(3*pow(T,2) + (-1 + u)*u*(1 + u)*(2 + u) + T*(-1 + 6*u*(1 + u)));
}

// equilibrium functions for h

__device__ inline double h0(double K, double T, double u, double v)
{
return (10*(16 + 3*K)*pow(T,3) + 3*T*(8 + 4*K - 40*pow(u,2) - 5*K*pow(u,2) + 20*pow(u,4) 
	+ K*pow(u,4) + (-5*(8 + K) + 4*(15 + K)*pow(u,2))*pow(v,2) + (20 + K)*pow(v,4)) 
	+ 30*pow(T,2)*(-4 + 9*pow(u,2) + 9*pow(v,2) + K*(-1 + pow(u,2) + pow(v,2))) 
	+ 3*(pow(u,2) + pow(v,2))*(4 + pow(u,4) - 5*pow(v,2) + pow(v,4) + pow(u,2)*(-5 + 4*pow(v,2))))/24.0;
}

__device__ inline double h1(double K, double T, double u, double v)
{
	return (-2*(16 + 3*K)*pow(T,3) - u*(1 + u)*(pow(u,2) + pow(v,2))*(-4 + pow(u,2) + 3*pow(v,2))
		- T*(u*(-4*(4 + K) - 4*(7 + K)*u + (14 + K)*pow(u,2) + (19 + K)*pow(u,3)) + (-4 + 3*u*(10
		+ K + (14 + K)*u))*pow(v,2) + 3*pow(v,4)) - pow(T,2)*(-16 + 6*u*(6 + 13*u) + 30*pow(v,2)
		+ K*(-4 + 6*u + 9*pow(u,2) + 3*pow(v,2))))/12.0;
}

__device__ inline double h5(double K, double T, double u, double v)
{
	return ((16 + 3*K)*pow(T,3) + 3*u*(1 + u)*v*(1 + v)*(pow(u,2) + pow(v,2)) + 3*pow(T,2)*((6 + K)*u 
		+ (9 + K)*pow(u,2) + v*(6 + K + (9 + K)*v)) + 3*T*(pow(u,3) + pow(u,4) + pow(v,3)*(1 + v)
		+ u*v*(6 + K + (9 + K)*v) + pow(u,2)*v*(9 + K + (12 + K)*v)))/24.0;
}

__device__ inline double h9(double K, double T, double u, double v)
{
	return ((16 + 3*K)*pow(T,3) + T*u*(-8 + K*(-1 + u)*(1 + u)*(2 + u) + u*(-7 + 2*u*(11 + 8*u))) 
		+ T*(-1 + 6*u*(1 + u))*pow(v,2) + (-1 + u)*u*(1 + u)*(2 + u)*(pow(u,2) + pow(v,2)) 
		+ pow(T,2)*(-4 + 36*u + 51*pow(u,2) + K*(-1 + 6*u*(1 + u)) + 3*pow(v,2)))/48.0;
}

////////////////////////////////////////////////////////////////////////////////
// Functions
////////////////////////////////////////////////////////////////////////////////

__device__ void
cuEq2D(double rho, double u_, double v_, double T_, double Tc, double R, double K, double* eqf, double* eqh)
{
	//returns the equilibrium values for each velocity vector given the current
	// macroscopic values and the corresponding reference quantities
	
	double sRTc = sqrt(R*Tc);
	double u = u_/sRTc;
	double v = v_/sRTc;
	double T = T_/Tc;

	eqf[0] = f0(rho,u,v,T);
	eqf[1] = f1(rho,u,v,T);
	eqf[2] = f1(rho,v,u,T);
	eqf[3] = f1(rho,-u,v,T);
	eqf[4] = f1(rho,-v,u,T);
	eqf[5] = f5(rho,u,v,T);
	eqf[6] = f5(rho,-u,v,T);
	eqf[7] = f5(rho,-u,-v,T);
	eqf[8] = f5(rho,u,-v,T);
	eqf[9] = f9(rho,u,v,T);
	eqf[10] = f9(rho,v,u,T);
	eqf[11] = f9(rho,-u,v,T);
	eqf[12] = f9(rho,-v,u,T);

	double rRTc = rho*R*Tc;

	eqh[0] = rRTc*h0(K,T,u,v);
	eqh[1] = rRTc*h1(K,T,u,v);
	eqh[2] = rRTc*h1(K,T,v,u);
	eqh[3] = rRTc*h1(K,T,-u,v);
	eqh[4] = rRTc*h1(K,T,-v,u);
	eqh[5] = rRTc*h5(K,T,u,v);
	eqh[6] = rRTc*h5(K,T,-u,v);
	eqh[7] = rRTc*h5(K,T,-u,-v);
	eqh[8] = rRTc*h5(K,T,u,-v);
	eqh[9] = rRTc*h9(K,T,u,v);
	eqh[10] =rRTc*h9(K,T,v,u);
	eqh[11] =rRTc*h9(K,T,-u,v);
	eqh[12] =rRTc*h9(K,T,-v,u);

	//check
	double sum_f = 0;
	double sum_h = 0;
	for (int i = 0; i < 13; i++)
	{
		sum_f += eqf[i];
		sum_h += eqh[i];
	}
	double U = sqrt(u_*u_ + v_*v_);
	double E = (U*U + (K+2.0)*R*T_)/2.0;

	double diff1 = sum_h/rho - E;
	double diff2 = sum_f - rho;

	if (diff1 > 1 || diff2 > 1)
	{
		errEq = 1;
	}
}

__device__ int
cuSign(double a)
{
	//returns: a = 0 -> 0, a = neg -> -1, a = pos -> 1
	if (a > 0)
	{
		return 1;
	}
	else if (a < 0)
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

__device__ void
cuMirIndex(int i, int mir_x, int mir_y, int& i_x, int& i_y)
{
	// gives the mirrored index about the axes defined by inputs mir_x, and mir_y

	// initialise with no mirror
	i_x = i;
	i_y = i;

	// mirror about x
	if (mir_x == 1)	
	{
		i_x = mirrorNS[i];
	}
	
	// mirror about y
	if(mir_y == 1)
	{
		i_y = mirrorEW[i];
	}

}

__device__
int cuIndex(int i, int pm, int d, simData* sim, int& mir)
{
	int Ny = (*sim).Ny;
	int Nx = (*sim).Nx;

	// d = 0 -> x
	// d = 1 -> y
	i = i + pm;
	mir = 0;

	if (d == 0)
	{
		if (i < 0)
		{
			if ((*sim).periodicX > 0)
			{
				i = Nx + i;
			}
			else if ((*sim).mirrorW > 0)
			{
				i = -i - 1;
				mir = 1;
			}
			else
			{
				i = 0; //zeroth order extrapolation
			}
		}
		else if (i > Nx - 1)
		{
			if ((*sim).periodicX > 0)
			{
				i = i - Nx;
			}
			else if ((*sim).mirrorE > 0)
			{
				i = 2*Nx - i - 1;
				mir = 1;
			}
			else
			{
				i = Nx - 1; //zeroth order extrapolation
			}
		}
	}

	else if (d == 1)
	{
		if (i < 0)
		{
			if ((*sim).periodicY > 0)
			{
				i = Ny + i;
			}
			else if ((*sim).mirrorS > 0)
			{
				i = -i - 1;
				mir = 1;
			}
			else
			{
				i = 0; //zeroth order extrapolation
			}
		}
		else if (i > Ny - 1)
		{
			if ((*sim).periodicY > 0)
			{
				i = i - Ny;
			}
			else if ((*sim).mirrorN > 0)
			{
				i = 2*Ny - i - 1;
				mir = 1;
			}
			else
			{
				i = Ny - 1; //zeroth order extrapolation
			}
		}
	}

	return i;
}

__device__ void
cuStencil(double* f_d, double* Sx, double* Sy, int n, int i, simData* sim, int* solid_d)
{
	// create a stencil of given length for the Flux method specified

	if (i == 0)
	{
		return;
	}

	int Ny = (*sim).Ny;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//global index
	int ix = blockIdx.x*blockDim.x + tx;
	int iy = blockIdx.y*blockDim.y + ty;

	// velocity vector, in integer values for indexing
	int ex = cuSign((*sim).e[i].x);
	int ey = cuSign((*sim).e[i].y);

	int low = (n - 1) / 2;	// max/min num of stencil from centre

	// calculate stencil along x and y
	int jx1 = ix - ex*low;	//start of stencil index
	int jy1 = iy - ey*low;

	int jx, jy;
	int mir_x, mir_y;
	int i_x, i_y;

	int sld_x = 0;
	int sld_y = 0;
	int sldx[5];
	int sldy[5];

	int sld;

	for (int j = 0; j < n; j++)	// put together stencil along axis lines, start on low side, go to hi side: [j-low <-> j+low]
	{
		jx = cuIndex(jx1,ex*j,0,sim,mir_y);
		jy = cuIndex(jy1,ey*j,1,sim,mir_x);

		cuMirIndex(i,mir_x,mir_y,i_x,i_y);	// mirror indexes if required

		sld = GSLD(ix,iy);

		// x
		Sx[j] = GF(jx,iy,i_y);

		if (sld >= 0)
		{
			sldx[j] = 1;
			sld_x = 1;
		}
		else
		{
			sldx[j] = 0;
		}

		// y
		Sy[j] = GF(ix,jy,i_x);

		if (sld >= 0)
		{
			sldy[j] = 1;
			sld_y = 1;
		}
		else
		{
			sldy[j] = 0;
		}
	}

	if (sld_x == 1)
	{
		int diff;	// difference between indexes of sldx
		int x_;		// location of edge solid node, outside coords
		int j_;		// location of edge solid node, stencil coords
		
		// find where edge solid node is
		for (int j = 0; j < n - 1; j++)
		{
			diff = sldx[j] - sldx[j+1];
			if (diff > 0)
			{
				j_ = j;
				x_ = cuIndex(jx1,ex*j,0,sim,mir_y);
				break;
			}
			else if (diff < 0)
			{
				j_ = j + 1;
				x_ = cuIndex(jx1,ex*(j + 1),0,sim,mir_y);
				break;
			}
		}

		// update solid part of stencil with linear extrapolation of actual stencil

		int xm1 = cuIndex(x_,ex*diff,0,sim,mir_y);		// closest fluid node
		int xm2 = cuIndex(x_,ex*2*diff,0,sim,mir_y);		// next closest fluid node

		Sx[j_] = 2*GF(xm1,iy,i) - GF(xm2,iy,i);

		int j__ = j_ - diff;

		if (j__ >= 0 && j__ < n) // update next one along if stencil is long enough
		{
			Sx[j__] = Sx[j_];
		}
	}

	if (sld_y == 1)
	{
		int diff;	// difference between indexes of sldx
		int y_;		// location of edge solid node, outside coords
		int j_;		// location of edge solid node, stencil coords
		
		// find where edge solid node is
		for (int j = 0; j < n - 1; j++)
		{
			diff = sldy[j] - sldy[j+1];
			if (diff > 0)
			{
				j_ = j;
				y_ = cuIndex(jy1,ey*j,1,sim,mir_x);
				break;
			}
			else if (diff < 0)
			{
				j_ = j + 1;
				y_ = cuIndex(jy1,ey*(j + 1),1,sim,mir_x);
				break;
			}
		}

		// update solid part of stencil with linear extrapolation of actual stencil

		int ym1 = cuIndex(y_,ey*diff,0,sim,mir_x);		// closest fluid node
		int ym2 = cuIndex(y_,ey*2*diff,0,sim,mir_x);		// next closest fluid node

		Sy[j_] = 2*GF(ix,ym1,i) - GF(ix,ym2,i);

		int j__ = j_ - diff;

		if (j__ >= 0 && j__ < n)	// update next one along if stencil is long enough
		{
			Sy[j__] = Sy[j_];
		}
	}
}

__device__ double
cuMinmod(double a, double b)
{
	// calculate minmod function

	double out;

	if ((abs(a) < abs(b)) && (a*b > 0))
		out = a;
	else if ((abs(b) < abs(a)) && (a*b > 0))
		out = b;
	else if (a == b)
		out = a;
	else if (a*b <= 0)
		out = 0;
	return out;
}

__device__ double
cuNND(double* S, double e)
{
	// calculate the flux by the NND method (2nd order accurate, CFL_max = 2/3)

	int n = 1;

	double Fp_I, Fp_Ip1, Fp_Im1;

	double vP = abs(e);

	if (vP == 0)
	{
		return 0;
	}

	Fp_I   = vP*S[n];
	Fp_Ip1 = vP*S[n+1];
	Fp_Im1 = vP*S[n-1];

	double dFp_Ip12, dFp_Im12;

	dFp_Ip12 = Fp_Ip1 - Fp_I;
	dFp_Im12 = Fp_I - Fp_Im1;

	double F_Ip12;

	F_Ip12 = Fp_I + 0.5*cuMinmod(dFp_Ip12,dFp_Im12);

	return F_Ip12;
}

__device__ double
cuWENO5(double* S, double e)
{
	// calculate the flux term of the WENO5 scheme for one flow direction

	double epsilon = (double) 1e-6;
	double posFlow = abs(e);
	double F_Ip12;

	if (posFlow == 0)
	{
		return 0;
	}
	else
	{
		double Sp[5];
		for (int i = 0; i < 5; i++)
		{
			Sp[i] = S[i]*posFlow;
		}

		int n = 2;

		double B0p, B1p, B2p, alpha0p, alpha1p, alpha2p;
		double omega0p, omega1p, omega2p, f0p, f1p, f2p;

		B0p = (13.0/12.0)*pow(Sp[n-2] - 2.0*Sp[n-1] + Sp[n],2) + (1.0/4.0)*pow(Sp[n-2] - 4.0*Sp[n-1] + 3.0*Sp[n],2);
		B1p = (13.0/12.0)*pow(Sp[n-1] - 2.0*Sp[n] + Sp[n+1],2) + (1.0/4.0)*pow(Sp[n-1] - Sp[n+1],2);
		B2p = (13.0/12.0)*pow(Sp[n] - 2.0*Sp[n+1] + Sp[n+2],2) + (1.0/4.0)*pow(3.0*Sp[n] - 4.0*Sp[n+1] + Sp[n+2],2);

		alpha0p = (1.0/10.0)*pow(1.0/(epsilon + B0p),2);
		alpha1p = (6.0/10.0)*pow(1.0/(epsilon + B1p),2);
		alpha2p = (3.0/10.0)*pow(1.0/(epsilon + B2p),2);

		omega0p = alpha0p/(alpha0p + alpha1p + alpha2p);
		omega1p = alpha1p/(alpha0p + alpha1p + alpha2p);
		omega2p = alpha2p/(alpha0p + alpha1p + alpha2p);

		f0p = (2.0/6.0)*Sp[n-2] - (7.0/6.0)*Sp[n-1] + (11.0/6.0)*Sp[n];
		f1p = -(1.0/6.0)*Sp[n-1] + (5.0/6.0)*Sp[n] + (2.0/6.0)*Sp[n+1];
		f2p = (2.0/6.0)*Sp[n] + (5.0/6.0)*Sp[n+1] - (1.0/6.0)*Sp[n+2];

		F_Ip12 = omega0p*f0p + omega1p*f1p + omega2p*f2p;
	}

	return F_Ip12;
}

__device__ void
cuFLUX(double* f, int i, simData* sim, int* solid_d, double2& flux)
{
	// choose flux method

	double ex, ey;

	ex = (*sim).e[i].x;
	ey = (*sim).e[i].y;

	if ((*sim).FMETHOD == 0)
	{
		//NND
		double Sx[3];
		double Sy[3];

		cuStencil(f, Sx, Sy, 3, i, sim, solid_d);

		flux.x = cuNND(Sx,ex);
		flux.y = cuNND(Sy,ey);
	}
	else if ((*sim).FMETHOD == 1)
	{
		// WENO5
		double Sx[5];
		double Sy[5];

		cuStencil(f, Sx, Sy, 5, i, sim, solid_d);

		flux.x = cuWENO5(Sx, ex);
		flux.y = cuWENO5(Sy, ey);
	}
}

__device__ double
cuCombineFLUX(double2* fluxOut, int i, simData* sim)
{
	// calculates the combined flux given all the fluxes out of each node

	int Ny = (*sim).Ny;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//global index
	int ix = blockIdx.x*blockDim.x + tx;
	int iy = blockIdx.y*blockDim.y + ty;

	// velocity vector, in integer values for indexing
	int ex = cuSign((*sim).e[i].x);
	int ey = cuSign((*sim).e[i].y);

	int mir_x, mir_y;

	int x_ = cuIndex(ix,-ex,0,sim,mir_y);
	int y_ = cuIndex(iy,-ey,1,sim,mir_x);

	if (i == 0)
	{
		return 0;
	}
	else
	{
		double dt = (*sim).dt;
		double dx = (*sim).dx;
		double dy = (*sim).dy;

		int i_x, i_y;

		cuMirIndex(i,mir_x,mir_y,i_x,i_y);	// mirror indexes if required


		double flux_out = dt*(GFOUTx(ix,iy,i)/dx + GFOUTy(ix,iy,i)/dy);

		double flux_in = dt*(GFOUTx(x_,iy,i_y)/dx + GFOUTy(ix,y_,i_x)/dy);

		return flux_out - flux_in;
	}
}

__device__ void
cuPosFlux(double* f, double* h, int i, int* solid_d, simData* sim, double2* f_flux, double2* h_flux)
{
	// function to calculate all positive fluxes along axis (x & y) lines for fluid nodes
	// store fluxes in global memory

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//global index
	int ix = blockIdx.x*blockDim.x + tx;
	int iy = blockIdx.y*blockDim.y + ty;

	int Ny = (*sim).Ny;

	// temporary flux storage

	double2 flux_f, flux_h;

	int sld = GSLD(ix,iy);

	if (sld < 0)	// only perform fluxes on fluid and permanent nodes
	{
		cuFLUX(f,i,sim,solid_d,flux_f);
		cuFLUX(h,i,sim,solid_d,flux_h);

		// store fluxes in global memory
		GFLUXFx(ix,iy,i) = flux_f.x;
		GFLUXFy(ix,iy,i) = flux_f.y;

		GFLUXHx(ix,iy,i) = flux_h.x;
		GFLUXHy(ix,iy,i) = flux_h.y;
	}
}

__global__ void
MACRO_PROPERTIES(double* f_d, double* h_d, simData* sim, double* rho_d, double2* u_d, double* T_d, double* p_d)
{

	// Block index

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	double rho, T, p, f;
	double2 u;
	double rho_ux, rho_uy, sum_h;

	rho = 0;
	rho_ux = 0;
	rho_uy = 0;
	sum_h = 0;

	//#pragma unroll 13
	for (int i = 0; i < 13; i++)
	{
		f = GF(ix,iy,i);

		rho += f;
		rho_ux += f*(*sim).e[i].x;
		rho_uy += f*(*sim).e[i].y;
		sum_h += GH(ix,iy,i);
	}

	u.x = rho_ux / rho;
	u.y = rho_uy / rho;

	double usq = sqrt(u.x*u.x + u.y*u.y);

	usq = usq*usq;

	T = 2.0*(sum_h/rho - usq/2.0)/((*sim).b*(*sim).R);

	p = rho*(*sim).R*T;

	// save macro-properties to device memory
	GRHO(ix,iy) = rho;
	GU(ix,iy) = u;
	GT(ix,iy) = T;
	GP(ix,iy) = p;
}

__global__ void
	GLOBAL_FLUXES(double* f, double* h, simData* sim, int* solid, double2* fluxf, double2* fluxh)
{
	// calculate all outgoing fluxes for the given distribution functions and save to global memory.

	__syncthreads();

	for (int i = 0; i < 13; i++)
	{
		// calc fluxes, save to global
		cuPosFlux(f, h, i, solid, sim, fluxf, fluxh);
	}
}

__global__ void 
WALL_FLUXES(double2* f_flux, double2* h_flux, int* solid_d, simData* sim, double* TW, double2* uW)
{
	// calculate fluxes into solid nodes and set fluxes out of solid to cancel them out

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//global index
	int ix = blockIdx.x*blockDim.x + tx;
	int iy = blockIdx.y*blockDim.y + ty;

	int Ny = (*sim).Ny;

	int xm1, ym1, i, inv_i;
	int mir_x, mir_y;

	double dx = (*sim).dx;
	double dy = (*sim).dy;

	double flux_in, flux_out, alpha;

	double feqW[13];
	double heqW[13];

	double ux, uy;

	int inx,iny;

	int sld = GSLD(ix,iy);

	if ( sld >= 0)	//wall nodes
	{
		flux_in = 0;
		flux_out = 0;

		cuEq2D(1.0, uW[sld].x, uW[sld].y, TW[sld], (*sim).Tc, (*sim).R, (*sim).K, feqW, heqW);

		// check surrounds for fluid node
		for (int j = 0; j < 4; j++)
		{
			xm1 = cuIndex(ix,xx[j],0,sim,mir_y);
			ym1 = cuIndex(iy,yy[j],1,sim,mir_x);

			int sld2 = GSLD(xm1,ym1);

			if (sld2 == -1) // if fluid
			{
				for (int k = 0; k < 4; k++)
				{
					i = ii[j][k];	// index of velocities into solid
					inv_i = inv[j][k];	// inverted velocities, out of solid

					// switches for turning off velocities that don't impinge on solid, or leave solid, through the cell pointed to by xx & yy
					inx = abs(xx[j]);	
					iny = abs(yy[j]);

					flux_in += inx*GFLUXFx(xm1,ym1,i)/dx + iny*GFLUXFy(xm1,ym1,i)/dy;		// flux into solid

					// absolute value of velocities
					ux = abs((*sim).e[inv_i].x);	
					uy = abs((*sim).e[inv_i].y);

					flux_out += inx*feqW[inv_i]*ux/dx + iny*feqW[inv_i]*uy/dy;				// flux out of solid, back along inverse velocity
				}
			}
		}

		alpha = flux_in/flux_out;	// correction factor to equalise flux in to flux out

		// load required fluxes into flux array
		for (int i = 0; i < 13; i++)
		{
			ux = abs((*sim).e[i].x);	
			uy = abs((*sim).e[i].y);

			 GFLUXFx(ix,iy,i) = alpha*ux*feqW[i];
			 GFLUXFy(ix,iy,i) = alpha*uy*feqW[i];

			 GFLUXHx(ix,iy,i) = alpha*ux*heqW[i];
			 GFLUXHy(ix,iy,i) = alpha*uy*heqW[i];
		}
	}
}

__global__ void
	RK1_STEP1_KERNEL(double* f_d, double* f1_d, double* h_d, double* h1_d, int* solid_d, double* TW, double2* uW, double* rho_d, double2* u_d, double* T_d, simData* sim)
{
	//perform RK1 stepping

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	double rho = GRHO(ix,iy);
	double2 u = GU(ix,iy);
	double T = GT(ix,iy);

	double feq[13];
	double heq[13];

	int sld;

	sld = GSLD(ix,iy);

	// --- RK STEP ONE----
	// calculate the first stage updated distribution functions 

	if (sld > -1 || sld == -2)	// check if node is solid or permanent, if it is, just propogate values
	{
		for (int i = 0; i < 13; i++)
		{
			GF1(ix,iy,i) = GF(ix,iy,i);
			GH1(ix,iy,i) = GH(ix,iy,i);
		}
	}
	else
	{
		// calculate relaxation times from macroscopic properties
		double tauf = (*sim).mu/(rho*(*sim).R*T);
		double tauh = tauf/(*sim).Pr;
		double tauhf = (tauh*tauf)/(tauf - tauh);

		double Tc = (*sim).Tc;
		double R = (*sim).R;
		double K = (*sim).K;

		cuEq2D(rho, u.x, u.y, T, Tc, R, K, feq, heq);

		double f1_, edotu;

		for (int i = 0; i < 13; i++)
		{
			// temp variables
			f1_ = (GF(ix,iy,i) + ((*sim).dt/tauf)*feq[i])/(1.0 + (*sim).dt/tauf);	//calc to temp variable first, for use later

			GF1(ix,iy,i) = f1_;		// save to global

			// temp variables
			edotu = (*sim).e[i].x*u.x + (*sim).e[i].y*u.y;

			GH1(ix,iy,i) = (GH(ix,iy,i) - (*sim).dt*edotu*((feq[i] - f1_)/tauhf) + ((*sim).dt/tauh)*heq[i])/(1.0 + (*sim).dt/tauh);
		}
	}
}

__global__ void
	RK1_COMBINE_KERNEL(double* f_d, double* f1_d, double2* fluxf1_d, double* h_d, double* h1_d, double2* fluxh1_d, int* solid_d, double* TW, double2* uW, double* rho_d, double2* u_d, double* T_d, simData* sim)
{
	//combine step of RK1

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	// macroscopic properties
	double rho = GRHO(ix,iy);
	double2 u = GU(ix,iy);
	double T = GT(ix,iy);

	// calculate relaxation times from macroscopic properties
	double tauf = (*sim).mu/(rho*(*sim).R*T);
	double tauh = tauf/(*sim).Pr;
	double tauhf = (tauh*tauf)/(tauf - tauh);

	double feq[13];
	double heq[13];

	double Tc = (*sim).Tc;
	double R = (*sim).R;
	double K = (*sim).K;

	cuEq2D(rho, u.x, u.y, T, Tc, R, K, feq, heq);

	double flux_f, flux_h, edotu;

	int sld = GSLD(ix,iy);

	for (int i = 0; i < 13; i++)
	{

		// --- RK COMBINATION ----
		// calculate the updated distribution functions
		if (sld == -1)
		{
			flux_f = cuCombineFLUX(fluxf1_d, i, sim);

			GF(ix,iy,i) = GF(ix,iy,i) - flux_f + ((*sim).dt/tauf)*(feq[i] - GF1(ix,iy,i));

			flux_h = cuCombineFLUX(fluxh1_d, i, sim);

			edotu = (*sim).e[i].x*u.x + (*sim).e[i].y*u.y;

			GH(ix,iy,i) = GH(ix,iy,i) - flux_h + ((*sim).dt/tauh)*(heq[i] - GH1(ix,iy,i)) - (((*sim).dt*edotu)/tauhf)*(feq[i] - GF1(ix,iy,i));
		}
	}
}

__global__ void
	RK3_STEP1_KERNEL(double* f_d, double* f1_d, double* h_d, double* h1_d, int* solid_d, double* TW, double2* uW, double* rho_d, double2* u_d, double* T_d, simData* sim)
{
	//perform RK3 stepping

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	double rho = GRHO(ix,iy);
	double2 u = GU(ix,iy);
	double T = GT(ix,iy);

	double feq[13];
	double heq[13];

	int sld = GSLD(ix,iy);

	// --- RK STEP ONE----
	// calculate the first stage updated distribution functions 

	if (sld > -1 || sld == -2)
	{
		for (int i = 0; i < 13; i++)
		{
			GF1(ix,iy,i) = GF(ix,iy,i);
			GH1(ix,iy,i) = GH(ix,iy,i);
		}
	}
	else
	{
		// calculate relaxation times from macroscopic properties
		double tauf = (*sim).mu/(rho*(*sim).R*T);
		double tauh = tauf/(*sim).Pr;
		double tauhf = (tauh*tauf)/(tauf - tauh);

		double Tc = (*sim).Tc;
		double R = (*sim).R;
		double K = (*sim).K;

		cuEq2D(rho, u.x, u.y, T, Tc, R, K, feq, heq);

		double f1_, edotu;

		for (int i = 0; i < 13; i++)
		{
			// temp variables
			f1_ = (GF(ix,iy,i) + ((*sim).dt/(2*tauf))*feq[i])/(1.0 + (*sim).dt/(2*tauf));	//calc to temp variable first, for use later

			GF1(ix,iy,i) = f1_;		// save to global

			// temp variables
			edotu = (*sim).e[i].x*u.x + (*sim).e[i].y*u.y;

			GH1(ix,iy,i) = (GH(ix,iy,i) - ((*sim).dt/2.0)*edotu*((feq[i] - f1_)/tauhf) + ((*sim).dt/(2.0*tauh))*heq[i])/(1.0 + (*sim).dt/(2.0*tauh));
		}
	}
}

__global__ void
	RK3_STEP2_KERNEL(double* f_d, double* f1_d, double* f2_d, double* h_d, double* h1_d, double* h2_d, int* solid_d, double* TW, double2* uW, double* rho_d, double2* u_d, double* T_d, simData* sim)
{
	//perform RK3 stepping

	// NOTE: as all equilibrium df are calulated from the same data, feq1 = feq2 etc, this allows for simplification of equations

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	double rho = GRHO(ix,iy);
	double2 u = GU(ix,iy);
	double T = GT(ix,iy);

	double feq[13];
	double heq[13];

	int sld = GSLD(ix,iy);

	// --- RK STEP TWO----
	// calculate the first stage updated distribution functions 

	if (sld > -1 || sld == -2)
	{
		for (int i = 0; i < 13; i++)
		{
			GF2(ix,iy,i) = GF(ix,iy,i);
			GH2(ix,iy,i) = GH(ix,iy,i);
		}
	}
	else
	{
		// calculate relaxation times from macroscopic properties
		double tauf = (*sim).mu/(rho*(*sim).R*T);
		double tauh = tauf/(*sim).Pr;
		double tauhf = (tauh*tauf)/(tauf - tauh);

		double Tc = (*sim).Tc;
		double R = (*sim).R;
		double K = (*sim).K;

		cuEq2D(rho, u.x, u.y, T, Tc, R, K, feq, heq);

		double f1_, f2_, edotu;

		for (int i = 0; i < 13; i++)
		{
			// temp variables
			f1_ = GF1(ix,iy,i);
			f2_ = (GF(ix,iy,i) + ((*sim).dt/(2*tauf))*f1_)/(1.0 + (*sim).dt/(2*tauf));	//calc to temp variable first, for use later

			GF2(ix,iy,i) = f2_;		// save to global

			// temp variables
			edotu = (*sim).e[i].x*u.x + (*sim).e[i].y*u.y;

			GH2(ix,iy,i) = (GH(ix,iy,i) - ((*sim).dt/2.0)*(edotu/tauhf)*(f1_ - f2_) + ((*sim).dt/2.0)*(GH1(ix,iy,i)/tauh))/(1.0 + (*sim).dt/(2.0*tauh));
		}
	}
}

__global__ void
	RK3_MACRO_PROPERTIES(double* f_d, double* f2_d, double2* fluxf2, double* h_d, double* h2_d, double2* fluxh2, simData* sim, double* rho3_d, double2* u3_d, double* T3_d)
{
	//perform RK3 step 3 macroscopic properties calculation

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//global index
	int ix = blockIdx.x*blockDim.x + tx;
	int iy = blockIdx.y*blockDim.y + ty;

	int Ny = (*sim).Ny;

	//macro variables
	__shared__ double rho_s[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ double2 u_s[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ double T_s[BLOCK_SIZE*BLOCK_SIZE];

	// initialise to zero
	SRHO(tx,ty) = 0;
	SU(tx,ty).x = 0;
	SU(tx,ty).y = 0;
	ST(tx,ty) = 0;

	double f, h, flux_f, flux_h;

	for (int i = 0; i < 13; i++)
	{
		// load f, and h, from global memory
		f = GF(ix,iy,i);
		h = GH(ix,iy,i);

		//calculate fluxes
		flux_f = cuCombineFLUX(fluxf2, i, sim);
		flux_h = cuCombineFLUX(fluxh2, i, sim);		

		SRHO(tx,ty) += f - flux_f;
		SU(tx,ty).x += (f - flux_f)*(*sim).e[i].x;
		SU(tx,ty).y += (f - flux_f)*(*sim).e[i].y;
		ST(tx,ty) += h - flux_h;		
	}
	
	SU(tx,ty).x = SU(tx,ty).x / SRHO(tx,ty);
	SU(tx,ty).y = SU(tx,ty).y / SRHO(tx,ty);

	double usq = sqrt(SU(tx,ty).x*SU(tx,ty).x + SU(tx,ty).y*SU(tx,ty).y);

	usq = usq*usq;
	
	ST(tx,ty) = 2.0*(ST(tx,ty)/SRHO(tx,ty) - usq/2.0)/((*sim).b*(*sim).R);

	// save to global memory
	GRHO3(ix,iy) = SRHO(tx,ty);
	GU3(ix,iy) = SU(tx,ty);
	GT3(ix,iy) = ST(tx,ty);
}

__global__ void
RK3_STEP3_KERNEL(double* f_d, double* f2_d, double*f3_d, double2* fluxf2, double* h_d, double* h2_d, double* h3_d, double2* fluxh2, int* solid_d, double* TW, double2* uW, 
double* rho_d, double2* u_d, double* T_d, double* rho3_d, double2* u3_d, double* T3_d, simData* sim)
{
	//perform RK3 stepping

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	double rho3 = GRHO3(ix,iy);
	double2 u3 = GU3(ix,iy);
	double T3 = GT3(ix,iy);
	
	double feq3[13];
	double heq3[13];

	double Tc = (*sim).Tc;
	double R = (*sim).R;
	double K = (*sim).K;

	int sld = GSLD(ix,iy);

	// --- RK STEP THREE ----
	// calculate the third stage updated distribution functions 

	// macroscopic properties
	double rho2 = GRHO(ix,iy);
	double2 u2 = GU(ix,iy);
	double T2 = GT(ix,iy);

	// calculate relaxation times from macroscopic properties
	double tauf2 = (*sim).mu/(rho2*(*sim).R*T2);
	double tauh2 = tauf2/(*sim).Pr;
	double tauhf2 = (tauh2*tauf2)/(tauf2 - tauh2);

	double tauf3 = (*sim).mu/(rho3*(*sim).R*T3);
	double tauh3 = tauf3/(*sim).Pr;
	double tauhf3 = (tauh3*tauf3)/(tauf3 - tauh3);

	double feq2[13];
	double heq2[13];

	cuEq2D(rho2, u2.x, u2.y, T2, Tc, R, K, feq2, heq2);
	cuEq2D(rho3, u3.x, u3.y, T3, Tc, R, K, feq3, heq3);

	double flux_f2, flux_h2, edotu2, edotu3, f2_, f3_;

	for (int i = 0; i < 13; i++)
	{
		// calculate the updated distribution functions
		if (sld == -1)
		{
			flux_f2 = cuCombineFLUX(fluxf2, i, sim);

			f2_ = GF2(ix,iy,i);

			f3_ = (GF(ix,iy,i) - flux_f2 + ((*sim).dt/(2*tauf2))*(feq2[i] - f2_) + ((*sim).dt/(2*tauf3))*feq3[i])/(1 + (*sim).dt/(2*tauf3));

			GF3(ix,iy,i) = f3_;

			flux_h2 = cuCombineFLUX(fluxh2, i, sim);

			edotu2 = (*sim).e[i].x*u2.x + (*sim).e[i].y*u2.y;
			edotu3 = (*sim).e[i].x*u3.x + (*sim).e[i].y*u3.y;

			GH3(ix,iy,i) = (GH(ix,iy,i) - flux_h2 - ((*sim).dt/2.0)*(edotu2*((feq2[i] - f2_)/tauhf2) + edotu3*((feq3[i] - f3_)/tauhf3)) + 
				((*sim).dt/2.0)*((heq2[i] - GH2(ix,iy,i))/tauh2) + ((*sim).dt/2.0)*(heq3[i]/tauh3))/(1 + (*sim).dt/(2*tauh3));
		}
		else
		{
			GF3(ix,iy,i) = GF(ix,iy,i);
			GH3(ix,iy,i) = GH(ix,iy,i);
		}
	}
}

__global__ void
RK3_COMBINE_KERNEL(double* f_d, double* f2_d, double* f3_d, double2* fluxf2, double2* fluxf3, double* h_d, double* h2_d, double* h3_d, 
double2* fluxh2, double2* fluxh3, double* rho_d, double2* u_d, double* T_d, double* rho3_d, double2* u3_d, double* T3_d, simData* sim, int* solid_d)
{
	//perform RK3 combination step

	//global index
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	int Ny = (*sim).Ny;

	// macroscopic properties
	double rho2 = GRHO(ix,iy);
	double2 u2 = GU(ix,iy);
	double T2 = GT(ix,iy);

	double rho3 = GRHO3(ix,iy);
	double2 u3 = GU3(ix,iy);
	double T3 = GT3(ix,iy);

	// calculate relaxation times from macroscopic properties
	double tauf2 = (*sim).mu/(rho2*(*sim).R*T2);
	double tauh2 = tauf2/(*sim).Pr;
	double tauhf2 = (tauh2*tauf2)/(tauf2 - tauh2);

	double tauf3 = (*sim).mu/(rho3*(*sim).R*T3);
	double tauh3 = tauf3/(*sim).Pr;
	double tauhf3 = (tauh3*tauf3)/(tauf3 - tauh3);

	double feq2[13];
	double heq2[13];

	double feq3[13];
	double heq3[13];

	double Tc = (*sim).Tc;
	double R = (*sim).R;
	double K = (*sim).K;

	cuEq2D(rho2, u2.x, u2.y, T2, Tc, R, K, feq2, heq2);
	cuEq2D(rho3, u3.x, u3.y, T3, Tc, R, K, feq3, heq3);

	double flux_f2, flux_h2, flux_f3, flux_h3, edotu2, edotu3;
	double f2_, f3_, h2_, h3_;

	int sld = GSLD(ix,iy);

	for (int i = 0; i < 13; i++)
	{
		// --- RK COMBINATION ----
		// calculate the updated distribution functions
		if (sld == -1)
		{

			f2_ = GF2(ix,iy,i);
			f3_ = GF3(ix,iy,i);

			flux_f2 = cuCombineFLUX(fluxf2, i, sim);
			flux_f3 = cuCombineFLUX(fluxf3, i, sim);

			GF(ix,iy,i) = GF(ix,iy,i) - (1.0/2.0)*(flux_f2 + flux_f3) + ((*sim).dt/2.0)*((feq2[i] - f2_)/tauf2 + (feq3[i] - f3_)/tauf3);

			h2_ = GH2(ix,iy,i);
			h3_ = GH3(ix,iy,i);

			flux_h2 = cuCombineFLUX(fluxh2, i, sim);
			flux_h3 = cuCombineFLUX(fluxh3, i, sim);

			edotu2 = (*sim).e[i].x*u2.x + (*sim).e[i].y*u2.y;
			edotu3 = (*sim).e[i].x*u3.x + (*sim).e[i].y*u3.y;

			GH(ix,iy,i) = GH(ix,iy,i) - 0.5*(flux_h2 + flux_h3) + ((*sim).dt/2.0)*((heq2[i] - h2_)/tauh2 + (heq3[i] - h3_)/tauh3) 
				- ((*sim).dt/2.0)*((edotu2/tauhf2)*(feq2[i] - f2_) + (edotu3/tauhf3)*(feq3[i] - f3_));
		}
	}
	__syncthreads();
}
